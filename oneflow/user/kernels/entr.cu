#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <hip/hip_math_constants.h>
#include "oneflow/user/kernels/entr.h"
#include "oneflow/core/common/device_type.pb.h"
#include "oneflow/user/kernels/elementwise_xpu_kernel.cuh"
namespace oneflow {
#ifdef WITH_CUDA
namespace {
template<typename T>
__inline__ __device__ T Nan();

template<>
__inline__ __device__ float Nan<float>() {
  return HIP_NAN_F;
}

template<>
__inline__ __device__ double Nan<double>() {
  return HIP_NAN;
}
}  // namespace
template<typename T>
struct EntrFunctor<DeviceType::kCUDA, T> {
  OF_DEVICE_FUNC T operator()(const T x) const {
    if (x > 0) {
      return -x * log(x);
    } else if (x == static_cast<T>(0)) {
      return static_cast<T>(0);
    } else {
      // -inf
      return -INFINITY;
    }
  }
};
template<typename T>
struct EntrGradFunctor<DeviceType::kCUDA, T> {
  OF_DEVICE_FUNC T operator()(const T x, const T dy) const {
    if (x > 0) {
      return (-log(x) - 1) * dy;
    } else if (x == static_cast<T>(0.0)) {
      // inf
      return INFINITY;
    } else {
      return Nan<T>();
    }
  }
};
REGISTER_ENTR_KERNEL_DEVICE_TYPE(DeviceType::kCUDA, float);
REGISTER_ENTR_KERNEL_DEVICE_TYPE(DeviceType::kCUDA, double);
#endif  // WITH_CUDA
}  // namespace oneflow
