#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/model_update_kernel_util.h"
#include "oneflow/user/kernels/indexed_slices_reduce_sum_kernel_util.h"
#include "oneflow/core/common/balanced_splitter.h"
#include "oneflow/core/kernel/cuda_graph_support.h"

namespace oneflow {

namespace {

// Refer from https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_apply.cuh
constexpr int depth_to_max_tensors[5] = {110, 64, 48, 36, 30};

template<typename T, typename G, int n>
struct TensorTupleParams {
  G* model_diff_addresses[depth_to_max_tensors[n - 1]];
  T* model_addresses[depth_to_max_tensors[n - 1]];
  int64_t sizes[depth_to_max_tensors[n - 1]];
};

// template<typename T, typename G, int n>
// __global__ void MultiTensorSGDUpdateGpu(int64_t num_tensor, T scale, const float l1, const float l2,
//                                         const float weight_decay, float learning_rate_val,
//                                         const float* learning_rate, const T* scale_by_ptr,
//                                         const int64_t* skip_if,
//                                         TensorTupleParams<T, G, n> meta_data) {
//   if (skip_if != nullptr && *skip_if != 0) { return; }
//   if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
//   if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }

//   for (int64_t tensor_idx = 0; tensor_idx < num_tensor; tensor_idx++) {
//     CUDA_1D_KERNEL_LOOP(i, meta_data.sizes[tensor_idx]) {
//       SGDUpdateFunctor<T, G>()(meta_data.model_diff_addresses[tensor_idx] + i,
//                                meta_data.model_addresses[tensor_idx] + i, scale, l1, l2,
//                                weight_decay, learning_rate_val);
//     }
//   }
// }


template<typename T, typename G, int n>
__global__ void MultiTensorSGDUpdateGpu(int64_t num_tensor, T scale, const float l1, const float l2,
                                        const float weight_decay, float learning_rate_val,
                                        const float* learning_rate, const T* scale_by_ptr,
                                        const int64_t* skip_if,
                                        TensorTupleParams<T, G, n> meta_data) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  int64_t v_block_id = blockIdx.x; 
  
  for (int64_t tensor_idx = 0; tensor_idx < num_tensor; tensor_idx++) {
    if(v_block_id == 0){
      for(int64_t i = v_block_id * blockDim.x + threadIdx.x; i < meta_data.sizes[tensor_idx]; i+= blockDim.x * gridDim.x)  {
        SGDUpdateFunctor<T, G>()(meta_data.model_diff_addresses[tensor_idx] + i,
                                meta_data.model_addresses[tensor_idx] + i, scale, l1, l2,
                                weight_decay, learning_rate_val);
      }
    } 
    const int64_t tensor_elem_cnt = meta_data.sizes[tensor_idx]; 
    const int64_t block_offset = ((tensor_elem_cnt + blockDim.x - 1) / blockDim.x) % gridDim.x;
    v_block_id -= block_offset; 
    if(v_block_id < 0) { v_block_id += gridDim.x; }
  }
}

template<DeviceType device_type, typename T, typename G>
class MultiTensorSGDUpdateKernel final : public user_op::OpKernel,
                                         public user_op::CudaGraphSupport {
 public:
  MultiTensorSGDUpdateKernel() = default;
  ~MultiTensorSGDUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int64_t n_tensor = ctx->input_size("model");
    const double scale = ctx->Attr<double>("scale");
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const float weight_decay = ctx->Attr<float>("weight_decay");
    const float* learning_rate_ptr = nullptr;
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");

    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), ctx->Tensor4ArgNameAndIndex("model", 0)->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }

    TensorTupleParams<T, G, 2> tensor_tuple_params{};
    int32_t count = 0;
    for (int i = 0; i < n_tensor; i++) {
      tensor_tuple_params.model_diff_addresses[count] =
          (ctx->Tensor4ArgNameAndIndex("model_diff", i))->mut_dptr<G>();
      tensor_tuple_params.model_addresses[count] =
          (ctx->Tensor4ArgNameAndIndex("model", i))->mut_dptr<T>();
      tensor_tuple_params.sizes[count] =
          (ctx->Tensor4ArgNameAndIndex("model", i))->shape().elem_cnt();
      count += 1;
      if (count == depth_to_max_tensors[1] || i == n_tensor - 1) {
        MultiTensorSGDUpdateGpu<T, G, 2>
            <<<16384, 256, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                count, static_cast<T>(scale), l1, l2, weight_decay, learning_rate_val,
                learning_rate_ptr, scale_by_ptr, skip_if_ptr, tensor_tuple_params);
        count = 0;
      }
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return true; }
};

#define REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(device, dtype, gtype)              \
  REGISTER_USER_KERNEL("multi_tensor_sgd_update")                                         \
      .SetCreateFn<MultiTensorSGDUpdateKernel<device, dtype, gtype>>()                    \
      .SetIsMatchedHob((user_op::HobDeviceType() == device)                               \
                       && (user_op::HobDataType("model", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("model_diff", 0) == GetDataType<gtype>::value));

REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, float, half);
REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, float, float);
REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, double, double);

}  // namespace

}  // namespace oneflow
