#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/model_update_kernel_util.h"
#include "oneflow/user/kernels/indexed_slices_reduce_sum_kernel_util.h"
#include "oneflow/core/common/balanced_splitter.h"
#include "oneflow/core/kernel/cuda_graph_support.h"

namespace oneflow {

namespace {

constexpr int kBlockSize = 256;

// Kernel arg size has 4K limit.
constexpr int max_tensors[5] = {160, 80, 40, 20, 15};

template<typename T, typename G, int n>
struct TensorTupleParams {
  G* model_diff_addresses[max_tensors[n - 1]];
  T* model_addresses[max_tensors[n - 1]];
  int64_t sizes[max_tensors[n - 1]];
  int64_t block_offset[max_tensors[n - 1]];
};

constexpr int kUnrollSize = 2; 

// template<typename T, typename G, int n>
// __global__ void MultiTensorSGDUpdateGpu(int64_t num_tensor, T scale, const float l1, const float l2,
//                                         const float weight_decay, float learning_rate_val,
//                                         const float* learning_rate, const T* scale_by_ptr,
//                                         const int64_t* skip_if,
//                                         TensorTupleParams<T, G, n> meta_data) {
//   if (skip_if != nullptr && *skip_if != 0) { return; }
//   if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
//   if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
//   int64_t v_block_id = blockIdx.x;
//   for (int64_t tensor_idx = 0; tensor_idx < num_tensor; tensor_idx++) {
//     for (int64_t i = v_block_id * blockDim.x + threadIdx.x; i < meta_data.sizes[tensor_idx];
//          i += blockDim.x * gridDim.x) {
//       SGDUpdateFunctor<T, G>()(meta_data.model_diff_addresses[tensor_idx] + i,
//                                meta_data.model_addresses[tensor_idx] + i, scale, l1, l2,
//                                weight_decay, learning_rate_val);
//     }
//     v_block_id -= meta_data.block_offset[tensor_idx];
//     if (v_block_id < 0) { v_block_id += gridDim.x; }
//   }
// }


// template<typename T, typename G, int n>
// __global__ void MultiTensorSGDUpdateGpu(int64_t num_tensor, T scale, const float l1, const float l2,
//                                         const float weight_decay, float learning_rate_val,
//                                         const float* learning_rate, const T* scale_by_ptr,
//                                         const int64_t* skip_if,
//                                         TensorTupleParams<T, G, n> meta_data) {
//   if (skip_if != nullptr && *skip_if != 0) { return; }
//   if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
//   if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
//   int64_t v_block_id = blockIdx.x;
//   for (int64_t tensor_idx = 0; tensor_idx < num_tensor; tensor_idx++) {
//     const int64_t tensor_elem_cnt = meta_data.sizes[tensor_idx];
//     for (int64_t i = v_block_id * blockDim.x * kUnrollSize + threadIdx.x; i < tensor_elem_cnt;
//          i += blockDim.x * gridDim.x * kUnrollSize) {
//       #pragma unroll
//       for(int32_t ilp = 0; ilp < kUnrollSize; ilp++){
//         int64_t actual_idx = i + ilp * v_block_id * blockDim.x; 
//         if(actual_idx < tensor_elem_cnt){
//           SGDUpdateFunctor<T, G>()(meta_data.model_diff_addresses[tensor_idx] + actual_idx,
//                                meta_data.model_addresses[tensor_idx] + actual_idx, scale, l1, l2,
//                                weight_decay, learning_rate_val);
//         }
//       }
//     }
//     v_block_id -= meta_data.block_offset[tensor_idx];
//     if (v_block_id < 0) { v_block_id += gridDim.x; }
//   }
// }


template<typename T, typename G, int n>
__global__ void MultiTensorSGDUpdateGpu(int64_t num_tensor, T scale, const float l1, const float l2,
                                        const float weight_decay, float learning_rate_val,
                                        const float* learning_rate, const T* scale_by_ptr,
                                        const int64_t* skip_if,
                                        TensorTupleParams<T, G, n> meta_data) {
  if (skip_if != nullptr && *skip_if != 0) { return; }
  if (learning_rate != nullptr) { learning_rate_val = *learning_rate; }
  if (scale_by_ptr != nullptr) { scale *= *scale_by_ptr; }
  int64_t v_block_id = blockIdx.x;
  for (int64_t tensor_idx = 0; tensor_idx < num_tensor; tensor_idx++) {
    const int64_t tensor_elem_cnt = meta_data.sizes[tensor_idx];
    for (int64_t i = v_block_id * blockDim.x * kUnrollSize + threadIdx.x; i < tensor_elem_cnt;
         i += blockDim.x * gridDim.x * kUnrollSize) {
      
      T model_val[kUnrollSize] = {0};
      T model_diff[kUnrollSize] = {0};

      #pragma unroll
      for(int32_t ilp = 0; ilp < kUnrollSize; ilp++){
        int64_t actual_idx = i + ilp * v_block_id * blockDim.x; 
        if(actual_idx < tensor_elem_cnt){
          model_val[ilp] = *(meta_data.model_addresses[tensor_idx] + actual_idx); 
          model_diff[ilp] = *(meta_data.model_diff_addresses[tensor_idx] + actual_idx); 
        }
      }

      #pragma unroll
      for(int32_t ilp = 0; ilp < kUnrollSize; ilp++){
        int64_t actual_idx = i + ilp * v_block_id * blockDim.x; 
        if(actual_idx < tensor_elem_cnt){
          model_diff[ilp] = CastScaleRegularizeGradientFunctor<T, G>()(model_diff[ilp], model_val[ilp], scale, l1, l2); 
          model_val[ilp] = model_val[ilp] - learning_rate_val * (model_diff[ilp] + weight_decay * model_val[ilp]); 
        }
      }

      #pragma unroll
      for(int32_t ilp = 0; ilp < kUnrollSize; ilp++){
        int64_t actual_idx = i + ilp * v_block_id * blockDim.x; 
        if(actual_idx < tensor_elem_cnt){
          *(meta_data.model_addresses[tensor_idx] + actual_idx) = model_val[ilp]; 
        }
      }

    }
    v_block_id -= meta_data.block_offset[tensor_idx];
    if (v_block_id < 0) { v_block_id += gridDim.x; }
  }
}


unsigned int ComputeGridSize(ep::Stream* stream, const int32_t block_size, const int64_t elem_cnt) {
  auto* hip_stream = stream->As<ep::CudaStream>();
  const int32_t max_threads_multi_process =
      hip_stream->device_properties().maxThreadsPerMultiProcessor;
  const int32_t multi_processor_count = hip_stream->device_properties().multiProcessorCount;
  unsigned int blocks_per_sm = max_threads_multi_process / block_size;
  unsigned int grid_size = ((elem_cnt + block_size - 1) / block_size);
  grid_size = std::min((unsigned int)multi_processor_count * blocks_per_sm, grid_size);
  return grid_size;
}

template<DeviceType device_type, typename T, typename G>
class MultiTensorSGDUpdateKernel final : public user_op::OpKernel,
                                         public user_op::CudaGraphSupport {
 public:
  MultiTensorSGDUpdateKernel() = default;
  ~MultiTensorSGDUpdateKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int64_t n_tensor = ctx->input_size("model");
    const double scale = ctx->Attr<double>("scale");
    const float l1 = ctx->Attr<float>("l1");
    const float l2 = ctx->Attr<float>("l2");
    const float weight_decay = ctx->Attr<float>("weight_decay");
    const float* learning_rate_ptr = nullptr;
    const float learning_rate_val = ctx->Attr<float>("learning_rate_val");

    if (ctx->has_input("learning_rate", 0)) {
      const user_op::Tensor* learning_rate = ctx->Tensor4ArgNameAndIndex("learning_rate", 0);
      learning_rate_ptr = learning_rate->dptr<float>();
    }
    const T* scale_by_ptr = nullptr;
    if (ctx->has_input("scale_by_tensor", 0)) {
      const user_op::Tensor* scale_by_tensor = ctx->Tensor4ArgNameAndIndex("scale_by_tensor", 0);
      CHECK_EQ(scale_by_tensor->data_type(), ctx->Tensor4ArgNameAndIndex("model", 0)->data_type());
      CHECK_EQ(scale_by_tensor->shape().elem_cnt(), 1);
      scale_by_ptr = scale_by_tensor->dptr<T>();
    }
    const int64_t* skip_if_ptr = nullptr;
    if (ctx->has_input("skip_if", 0)) {
      const user_op::Tensor* skip_if = ctx->Tensor4ArgNameAndIndex("skip_if", 0);
      CHECK_EQ(skip_if->shape().elem_cnt(), 1);
      skip_if_ptr = skip_if->dptr<int64_t>();
    }

    TensorTupleParams<T, G, 2> tensor_tuple_params{};
    int32_t count = 0;
    int32_t total_elem_cnt = 0;
    for (int tensor_idx = 0; tensor_idx < n_tensor; tensor_idx++) {
      tensor_tuple_params.model_diff_addresses[count] =
          (ctx->Tensor4ArgNameAndIndex("model_diff", tensor_idx))->mut_dptr<G>();
      tensor_tuple_params.model_addresses[count] =
          (ctx->Tensor4ArgNameAndIndex("model", tensor_idx))->mut_dptr<T>();
      const int64_t tensor_elem_cnt =
          ctx->Tensor4ArgNameAndIndex("model", tensor_idx)->shape().elem_cnt();
      tensor_tuple_params.sizes[count] = tensor_elem_cnt;

      count += 1;
      total_elem_cnt += tensor_elem_cnt;
      if (count == max_tensors[1] || tensor_idx == n_tensor - 1) {
        const unsigned int grid_size =
            ComputeGridSize(ctx->stream()->As<ep::CudaStream>(), kBlockSize, total_elem_cnt);
        for (int i = 0; i < count; i++) {
          tensor_tuple_params.block_offset[i] =
              ((tensor_tuple_params.sizes[i] + kBlockSize * kUnrollSize - 1) / (kBlockSize * kUnrollSize)) % grid_size;
        }
        MultiTensorSGDUpdateGpu<T, G, 2>
            <<<grid_size, kBlockSize, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
                count, static_cast<T>(scale), l1, l2, weight_decay, learning_rate_val,
                learning_rate_ptr, scale_by_ptr, skip_if_ptr, tensor_tuple_params);
        count = 0;
        total_elem_cnt = 0;
      }
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return true; }
};

#define REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(device, dtype, gtype)              \
  REGISTER_USER_KERNEL("multi_tensor_sgd_update")                                         \
      .SetCreateFn<MultiTensorSGDUpdateKernel<device, dtype, gtype>>()                    \
      .SetIsMatchedHob((user_op::HobDeviceType() == device)                               \
                       && (user_op::HobDataType("model", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("model_diff", 0) == GetDataType<gtype>::value));

REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, float, half);
REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, float, float);
REGISTER_MULTI_TENSOR_UPDATE_SGD_UPDATE_KERNEL(DeviceType::kCUDA, double, double);

}  // namespace

}  // namespace oneflow
