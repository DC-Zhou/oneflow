#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"


namespace oneflow{

namespace {

// TODO add AFFINE STORE

template<typename SRC, typename DST, bool affine>
struct AffineStore{
    AffineStore(DST* y, int64_t row_size, int64_t channel_size, int64_t spatial_size, const DST* gamma, const DST* beta)
    : y(y), row_size(row_size), channel_size(channel_size), spatial_size(spatial_size), gamma(gamma), beta(beta) {}

    template<int PackSize>
    __device__ void store(const SRC* src, int64_t row, int64_t col){
        cuda::layer_norm::Pack<DST, PackSize> y_pack;
        const int64_t offset = row * row_size + col; 
        const int64_t packed_offset = offset / PackSize;
        const int64_t gamma_beta_offset = (offset / spatial_size) % channel_size;
        DST gamma_val = gamma[gamma_beta_offset]; 
        DST beta_val = beta[gamma_beta_offset]; 

    #pragma unroll
        for (int i = 0; i < PackSize; ++i) {
            DST normalized_i = static_cast<DST>(src[i]);
            if(affine){
                y_pack.elem[i] = normalized_i * gamma_val + beta_val;
            } else {
                // Direct Store. 
                y_pack.elem[i] = normalized_i; 
            }
        }
        *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + packed_offset) = y_pack.storage;
    }

    DST* y;
    int64_t row_size;
    int64_t channel_size;
    int64_t spatial_size;
    const DST* gamma;
    const DST* beta;
}; 

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormWarpImpl(hipStream_t stream, 
                                             LOAD load, STORE store,
                                             const int64_t rows, const int64_t cols,
                                             const int64_t spatial_size, 
                                             const double epsilon, ComputeType* mean,
                                             ComputeType* inv_variance) {
  return DispatchGroupNormWarpImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}


template<typename LOAD, typename STORE, typename ComputeType>
struct TryDispatchGroupNormBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance, bool* success) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t TryDispatchGroupNormBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                     const int64_t rows, const int64_t cols, 
                                                     const int64_t spatial_size, 
                                                     const double epsilon, ComputeType* mean,
                                                     ComputeType* inv_variance, bool* success) {
  return TryDispatchGroupNormBlockSMemImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance, success);
}

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                      const int64_t rows, const int64_t cols,
                                                      const int64_t spatial_size, 
                                                      const double epsilon, ComputeType* mean,
                                                      ComputeType* inv_variance) {
  return DispatchGroupNormBlockUncachedImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}


template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                  ComputeType* inv_variance) {
  if (cols <= 1024) {
    return DispatchGroupNormWarpImpl<LOAD, STORE, ComputeType>(stream, load, store, rows, cols, spatial_size, 
                                                               epsilon, mean, inv_variance);
  } else {
    // TODO
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchGroupNormBlockSMemImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance,
          &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchGroupNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
    }
    return hipSuccess;
  }
}

template<typename T, bool affine>
void GroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances, 
                        const int64_t norm_size, 
                        const int64_t channel_size, 
                        const int64_t spatial_size, 
                        const double epsilon, const T* x_ptr, const T* gamma_ptr,
                        const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                        user_op::Tensor* inv_variance) {
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    cuda::layer_norm::DirectLoad<T, ComputeType> load(x_ptr, norm_size);
    AffineStore<ComputeType, T, affine> store(y_ptr, norm_size, channel_size, spatial_size, gamma_ptr, beta_ptr);

    DispatchGroupNorm<decltype(load), decltype(store), ComputeType>(
        stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, spatial_size, 
        epsilon, mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchGroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, 
                                 const int64_t channel_size, 
                                 const int64_t spatial_size, 
                                 const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    GroupNormForwardGpu<T, true>(stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr, gamma_ptr,
                                       beta_ptr, y_ptr, mean, inv_variance);
  } else {
    GroupNormForwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr,
                                         gamma_ptr, beta_ptr, y_ptr, mean, inv_variance);
  }
}

} // namespace 

template<typename T>
class GroupNormGpuKernel final : public user_op::OpKernel{

public: 
    GroupNormGpuKernel() = default; 
    ~GroupNormGpuKernel() = default; 

private: 
    using user_op::OpKernel::Compute; 
    bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
    void Compute(user_op::KernelComputeContext* ctx) const override {
        const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0); 
        user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0); 
        user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
        user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
        const double epsilon = ctx->Attr<double>("epsilon");
        const int32_t num_groups = ctx->Attr<int32_t>("num_groups"); 
        CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
        const int64_t num_instances = mean->shape().elem_cnt();  // N*num_groups
        const int64_t norm_size = x->shape().elem_cnt() / num_instances;
        const int64_t batch_size = x->shape().At(0); 
        const int64_t channel_size = x->shape().At(1); 
        const int64_t spatial_size = x->shape().elem_cnt() / batch_size / channel_size; 
        printf("B x num_groups = %d \n", batch_size*num_groups); 
        printf("num instance is: %d \n", num_instances); 
        printf("Spatial size is: %d \n", spatial_size); 
        printf("CHannel size is: %d \n", channel_size); 
        const T* gamma_ptr = nullptr;
        const T* beta_ptr = nullptr;
        if (ctx->has_input("gamma", 0) && ctx->has_input("beta", 0)) {
          const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
          gamma_ptr = gamma->dptr<T>();
          CHECK_EQ(gamma->shape().elem_cnt(), channel_size);
          const user_op::Tensor* beta = ctx->Tensor4ArgNameAndIndex("beta", 0); 
          beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>();
          CHECK_EQ(beta->shape().elem_cnt(), channel_size);
        }
        DispatchGroupNormForwardGpu<T>(ctx->stream(), 
                                       num_instances, norm_size, 
                                       channel_size, 
                                       spatial_size, epsilon, 
                                       x->dptr<T>(),
                                       gamma_ptr, beta_ptr, y->mut_dptr<T>(), mean, inv_variance);

    }

}; 

#define REGISTER_GROUP_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("group_norm")                                   \
      .SetCreateFn<GroupNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

// REGISTER_GROUP_NORM_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_CUDA_KERNEL(float)
// REGISTER_GROUP_NORM_CUDA_KERNEL(double)

} // namespace oneflow 