#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace user_op {

namespace {

constexpr int32_t kBlockSize = 1024;

template<typename T>
__global__ void FusedBinaryCrossEntropyWithLogitsReduceMeanKernel(const T* input, const T* target,
                                                                  T* out, const int32_t elem_cnt) {
  T zero = static_cast<T>(0.0);
  T one = static_cast<T>(1.0);
  using BlockReduce = hipcub::BlockReduce<T, kBlockSize>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  T reduce_sum = 0.0;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const T input_val = input[i];
    const T target_val = target[i];
    const T max_val = -input_val < zero ? zero : -input_val;
    const T result =
        (one - target_val) * input_val + max_val + (log(exp(-max_val) + exp(-input_val - max_val)));
    reduce_sum += result;
  }
  const T block_reduce_sum = BlockReduce(temp_storage).Sum(reduce_sum);
  if (threadIdx.x == 0) { out[0] = block_reduce_sum / elem_cnt; }
}

template<>
__global__ void FusedBinaryCrossEntropyWithLogitsReduceMeanKernel<half>(const half* input,
                                                                        const half* target,
                                                                        half* out,
                                                                        const int32_t elem_cnt) {
  float zero = static_cast<float>(0.0);
  float one = static_cast<float>(1.0);
  using BlockReduce = hipcub::BlockReduce<float, kBlockSize>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  float reduce_sum = 0.0;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const float input_val = __half2float(input[i]);
    const float target_val = __half2float(target[i]);
    const float max_val = -input_val < zero ? zero : -input_val;
    const float result =
        (one - target_val) * input_val + max_val + (log(exp(-max_val) + exp(-input_val - max_val)));
    const float block_reduce_sum = BlockReduce(temp_storage).Sum(result);
    if (threadIdx.x == 0) { reduce_sum += block_reduce_sum; }
  }
  if (threadIdx.x == 0) { out[0] = __float2half(reduce_sum / elem_cnt); }
}

template<typename T>
__device__ __forceinline__ T CalSigmoid(const T x) {
  const T half_of_one = static_cast<T>(0.5);
  return half_of_one * tanh(half_of_one * x) + half_of_one;
}

template<>
__device__ __forceinline__ float CalSigmoid(const float x) {
  const float half_of_one = static_cast<float>(0.5);
  return half_of_one * tanhf(half_of_one * x) + half_of_one;
}

template<>
__device__ __forceinline__ half CalSigmoid(const half x) {
  return __float2half(CalSigmoid(__half2float(x)));
}

template<typename T>
struct BinaryCrossEntropyWithLogitsReduceMeanGradFunctor {
  OF_DEVICE_FUNC explicit BinaryCrossEntropyWithLogitsReduceMeanGradFunctor(
      const T elem_cnt_reciprocal, const T dy)
      : elem_cnt_reciprocal(elem_cnt_reciprocal), dy(dy) {}
  __device__ T operator()(const T input_val, const T target_val) const {
    return (CalSigmoid(input_val) - target_val) * dy * elem_cnt_reciprocal;
  }
  const T dy;
  const T elem_cnt_reciprocal;
};

template<typename T>
struct BinaryCrossEntropyWithLogitsReduceMeanGradDyptrFunctor {
  OF_DEVICE_FUNC explicit BinaryCrossEntropyWithLogitsReduceMeanGradDyptrFunctor(
      const int32_t elem_cnt, const T* dy_ptr)
      : elem_cnt_reciprocal(1.0f / elem_cnt), dy_ptr(dy_ptr) {}
  __device__ BinaryCrossEntropyWithLogitsReduceMeanGradFunctor<T> operator()() const {
    return BinaryCrossEntropyWithLogitsReduceMeanGradFunctor<T>(elem_cnt_reciprocal, *dy_ptr);
  }
  const T* dy_ptr;
  const T elem_cnt_reciprocal;
};

template<typename T>
class BinaryCrossEntropyWithLogitsMeanKernel final : public user_op::OpKernel {
 public:
  BinaryCrossEntropyWithLogitsMeanKernel() = default;
  ~BinaryCrossEntropyWithLogitsMeanKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* input_blob = ctx->Tensor4ArgNameAndIndex("input", 0);
    const auto* target_blob = ctx->Tensor4ArgNameAndIndex("target", 0);
    auto* out_blob = ctx->Tensor4ArgNameAndIndex("out", 0);

    const int64_t elem_cnt = input_blob->shape_view().elem_cnt();

    const T* input = input_blob->dptr<T>();
    const T* target = target_blob->dptr<T>();
    T* out = out_blob->mut_dptr<T>();

    FusedBinaryCrossEntropyWithLogitsReduceMeanKernel<<<
        1, kBlockSize, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        input_blob->dptr<T>(), target_blob->dptr<T>(), out_blob->mut_dptr<T>(),
        input_blob->shape_view().elem_cnt());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class BinaryCrossEntropyWithLogitsReduceMeanGradKernel final : public user_op::OpKernel {
 public:
  BinaryCrossEntropyWithLogitsReduceMeanGradKernel() = default;
  ~BinaryCrossEntropyWithLogitsReduceMeanGradKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* input_blob = ctx->Tensor4ArgNameAndIndex("input", 0);
    const auto* target_blob = ctx->Tensor4ArgNameAndIndex("target", 0);
    const auto* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    auto* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);

    const int64_t elem_cnt = input_blob->shape_view().elem_cnt();
    const T* dy = dy_blob->dptr<T>();
    const T* input = input_blob->dptr<T>();
    const T* target = target_blob->dptr<T>();
    T* dx = dx_blob->mut_dptr<T>();

    OF_CUDA_CHECK((cuda::elementwise::BinaryWithFactory(
        BinaryCrossEntropyWithLogitsReduceMeanGradDyptrFunctor<T>(elem_cnt, dy), elem_cnt, dx,
        input, target, ctx->stream()->As<ep::CudaStream>()->hip_stream())));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

}  // namespace

#define REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_KERNEL(dtype)                            \
  REGISTER_USER_KERNEL("binary_cross_entropy_with_logits_reduce_mean")                     \
      .SetCreateFn<BinaryCrossEntropyWithLogitsMeanKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("input", 0) == GetDataType<dtype>::value)  \
                       && (user_op::HobDataType("target", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

#define REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_GRAD_KERNEL(dtype)                       \
  REGISTER_USER_KERNEL("binary_cross_entropy_with_logits_reduce_mean_grad")                \
      .SetCreateFn<BinaryCrossEntropyWithLogitsReduceMeanGradKernel<dtype>>()              \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("input", 0) == GetDataType<dtype>::value)  \
                       && (user_op::HobDataType("target", 0) == GetDataType<dtype>::value) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value)     \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_KERNEL(half)
REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_KERNEL(float)
REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_KERNEL(double)

REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_GRAD_KERNEL(half)
REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_GRAD_KERNEL(float)
REGISTER_BINARY_CROSS_ENTROPY_REDUCE_MEAN_GRAD_KERNEL(double)

}  // namespace user_op
}  // namespace oneflow
